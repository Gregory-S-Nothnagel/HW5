/*

Gregory Nothnagel
gsnothnagel@crimson.ua.edu
CS 581
Homework #5

To Compile CUDA.cu game of life program, run the following script:

        #!/bin/bash

		source /apps/profiles/modules_asax.sh.dyn

		module load cuda/11.7.0

		nvcc CUDA.cu -o CUDA

To Run, use the following command (inside of a .sh file obviously) and run with run_gpu on class queue:

        ./CUDA <BOARD_SIZE> <MAX_ITERATIONS> <BOARD_OUTPUT_PATH>

For more details, see "Compilation" section of README.md in the github repo

* Then you can inspect the board output file to confirm that everything looks as expected

*/

#include <stdio.h>
#include <stdlib.h>  // For atoi
#include <hip/hip_runtime.h>

// Function to write the final grid to a file
void write_grid_to_file(const char *filename, int *grid, int width, int height) {
    FILE *file = fopen(filename, "w");
    if (!file) {
        perror("Error opening file");
        return;
    }

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            fprintf(file, "%d", grid[y * width + x]);
        }
        fprintf(file, "\n");
    }
        fprintf(file, "\n");

    fclose(file);
    printf("Final grid state written to %s\n", filename);
}

// Kernel to update the grid
__global__ void game_of_life(int *grid, int *newGrid, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure the thread is within the grid bounds
    if (x >= width || y >= height) return;


        int live_neighbors = 0;

    // Count live neighbors with boundary checks
    for (int nx = x - 1; nx <= x + 1; nx++) {
        for (int ny = y - 1; ny <= y + 1; ny++) {
            if (nx != x || ny != y) {
                                // Only count valid neighbors within the grid
                                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                                        live_neighbors += grid[ny * width + nx];
                                }
                        }
        }
    }

    // Apply Conway's rules
    int current = grid[y * width + x];

        if ((current && live_neighbors < 2) || (current && live_neighbors > 3) || (!current && live_neighbors == 3)) {
                newGrid[y * width + x] = !current;
        }
    else newGrid[y * width + x] = current;

        return;

}

// Main program
int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <grid size (N)> <max iterations> <output file>\n", argv[0]);
        return -1;
    }

    int N = atoi(argv[1]);
    int MAX_ITERATIONS = atoi(argv[2]);
    const char *outputFile = argv[3];

    if (N <= 0 || MAX_ITERATIONS <= 0) {
        printf("Both grid size and max iterations must be positive integers.\n");
        return -1;
    }

    const int width = N;
    const int height = N;

    size_t gridSize = width * height * sizeof(int);

    // Allocate memory for the grid on the host
    int *h_grid = (int *)malloc(gridSize);
    int *h_newGrid = (int *)malloc(gridSize);

    // Initialize the grid (random or predefined pattern)
    for (int i = 0; i < width * height; i++) {
        h_grid[i] = rand() % 2; // Randomly initialize cells as 0 or 1
    }

    // Allocate memory for the grid on the device
    int *d_grid, *d_newGrid;
    hipMalloc(&d_grid, gridSize);
    hipMalloc(&d_newGrid, gridSize);

    // Copy the initial grid to the device
    hipMemcpy(d_grid, h_grid, gridSize, hipMemcpyHostToDevice);

    // Define the execution configuration
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);


        int *currentGrid = d_grid;
        int *nextGrid = d_newGrid;

        // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording
    hipEventRecord(start, 0);

        for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
                game_of_life<<<blocksPerGrid, threadsPerBlock>>>(currentGrid, nextGrid, width, height);
                hipDeviceSynchronize();

                // Swap the grids
                int *temp = currentGrid;
                currentGrid = nextGrid;
                nextGrid = temp;
        }

        // Stop recording
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Clean up events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy the final grid back to the host
    hipMemcpy(h_grid, d_grid, gridSize, hipMemcpyDeviceToHost);

    // Write the final grid to the specified file
    write_grid_to_file(outputFile, h_grid, width, height);

    // Free memory
    free(h_grid);
    free(h_newGrid);
    hipFree(d_grid);
    hipFree(d_newGrid);

        printf("Time taken for %d iterations: %.3f ms\n", MAX_ITERATIONS, milliseconds);

    return 0;
}